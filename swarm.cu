#include "hip/hip_runtime.h"
#define GLM_FORCE_RADIANS

#include <iostream>

#include <hip/hip_runtime.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/norm.hpp>
#include <glm/gtx/vector_angle.hpp>

const float kRepulsionZoneRadius = 20.0;
const float kOrientationZoneRadius = 90.0;
const float kVisualFieldAngle = 180.0 / 180.0 * 3.14159;

#define ATTRACTION_WEIGHT 1.0f
#define ORIENTATION_WEIGHT 1.0f

const float epsilon = 1.0e-4;

// __device__ Functions

__constant__ glm::vec4 plane1;
__constant__ glm::vec4 plane2;
__constant__ glm::vec4 plane3;
__constant__ glm::vec4 plane4;
__constant__ glm::vec4 plane5;
__constant__ glm::vec4 plane6;

__device__ int GPU_globalindex() {
        return  blockIdx.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.x * blockDim.z * blockDim.y * blockDim.x +
                threadIdx.z * blockDim.y * blockDim.x +
                threadIdx.y * blockDim.x +
                threadIdx.x;
}

// Returns true iff the point b is inside a's field of view
__device__ bool is_in_visual_field(glm::vec3 a, glm::vec3 b) {
    return glm::angle(glm::normalize(a), glm::normalize(b)) <= kVisualFieldAngle / 2 + epsilon;
}

__device__ void closest_neighbors(int*& points_indices, int& n_points,
        int global_index, int total_number,
        glm::vec3 *positions, glm::vec3 *directions,
        float radius) {
  n_points = 0;
  for(int j=0;j<total_number;++j){
    if(j != global_index){
      if(glm::distance(positions[j], positions[global_index]) < radius
         && is_in_visual_field(directions[global_index], positions[j] - positions[global_index])){
        points_indices[n_points] = j;
        n_points++;
      }
    }
  }
}


__device__ glm::vec3 resultant(glm::vec3* positions, int* points_indices, int n_points, int cur_boid_index){
    glm::vec3 new_vec(0,0,0);
    for(int j=0;j<n_points;++j)
        new_vec += glm::normalize(positions[points_indices[j]] - positions[cur_boid_index]);
    return new_vec;
}

__device__ glm::vec3 resultant_direction(glm::vec3* directions, int* points_indices, int n_points, int cur_boid_index){
    glm::vec3 new_vec(0,0,0);
    for(int j=0;j<n_points;++j)
        new_vec += glm::normalize(directions[points_indices[j]]);
    return new_vec;
}

// __global__ Functions

__global__ void GPU_update_vector(glm::vec3 *directions_output, glm::vec3 *directions_input, glm::vec3 *positions, int num_boids){
        int index = GPU_globalindex();
        if(index < num_boids)
        {

                // directions_output[index] = glm::vec3(0,0,50);
                directions_output[index] = glm::vec3(0,0,0);
                int n_points = 0;
                int* points_indices = new int[num_boids];

                closest_neighbors(points_indices, n_points, index,
                        num_boids, positions, directions_input, kRepulsionZoneRadius);
                glm::vec3 sum_vector = resultant(positions, points_indices, n_points, index);
                if (n_points) {
                    // since we have neighbors the repulsion behavior is applied
                    directions_output[index] = -sum_vector;
                } else {
                    // if there aren't any neighbors in the repulsion zone
                    // we need to explore the orientation zone
                    closest_neighbors(points_indices, n_points, index, num_boids,
                            positions, directions_input, kOrientationZoneRadius);
                    sum_vector = resultant(positions, points_indices, n_points, index);
                    glm::vec3 sum_direction =
                        resultant_direction(directions_input, points_indices, n_points, index);
                    if (n_points) {
                        directions_output[index] =
                            sum_vector * ATTRACTION_WEIGHT + sum_direction * ORIENTATION_WEIGHT;
                    } else {
                        // SPECIAL CASE
                        // If there aren't any neighbors in any zone
                        // then keep the current direction
                        directions_output[index] = directions_input[index];
                    }
                }
                delete[] points_indices;
        }
}

__global__ void GPU_Update(glm::mat4 *modelMatrices, glm::vec3 *directions,
        glm::vec3 *updated_directions, glm::vec3 *positions,
        glm::vec3 *raxis, float *angles, int num_boids, float cT) {
        int i = GPU_globalindex();
        printf("%d\n", 2.0f);
        if(i < num_boids)
        {

                // float theta = 0.0;
                // glm::vec3 cr(0,0,0);
                // if(glm::length(directions[i]-updated_directions[i]) > epsilon)
                // {
                //         theta = glm::acos(glm::dot(glm::normalize(directions[i]),glm::normalize(updated_directions[i])));
                //         cr = glm::normalize(glm::cross(directions[i],updated_directions[i]));
                // }

                // if(glm::length(raxis[i]) > epsilon)
                // {
                //         modelMatrices[i] = glm::rotate(modelMatrices[i], -angles[i], raxis[i]);
                // }
                modelMatrices[i] = glm::translate(modelMatrices[i], updated_directions[i] * 0.000035f);
                // if(glm::length(raxis[i]) > epsilon)
                // {
                //         modelMatrices[i] = glm::rotate(modelMatrices[i], angles[i], raxis[i]);
                // }


                // if(glm::length(cr) > epsilon)
                // {
                //         modelMatrices[i] = glm::rotate(modelMatrices[i], theta, cr);
                //         raxis[i] = glm::normalize(glm::cross(glm::vec3(0.0,0.0,0.25),updated_directions[i]));
                //         angles[i] = glm::acos(glm::dot(glm::normalize(updated_directions[i]),glm::normalize(glm::vec3(0.0,0.0,0.25))));;
                // }

                // TODO Remove this line, we perform hipMemcpy later in the code
                directions[i] = updated_directions[i];
                positions[i] += directions[i] * 0.000035f;

        }
}

void update(glm::mat4 *modelMatrices, glm::vec3 *directions,
            glm::vec3 *updated_directions, glm::vec3 *positions,
            glm::vec3 *raxis, float *angles, int num_boids, float cT) {
        glm::mat4 *d_modelMatrices;
        glm::vec3 *new_directions, *new_updated_directions, *new_positions, *new_raxis;
        float *new_angles;

        size_t m4size = num_boids * sizeof(glm::mat4);
        size_t v3size = num_boids * sizeof(glm::vec3);
        size_t fsize  = num_boids * sizeof(float);

        hipMalloc(&d_modelMatrices, m4size);
        hipMalloc(&new_directions, v3size);
        hipMalloc(&new_updated_directions, v3size);
        hipMalloc(&new_positions, v3size);
        hipMalloc(&new_raxis, v3size);
        hipMalloc(&new_angles, fsize);

        hipMemcpy(d_modelMatrices, modelMatrices, m4size, hipMemcpyHostToDevice);
        hipMemcpy(new_directions, directions, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_updated_directions, updated_directions, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_positions, positions, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_raxis, raxis, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_angles, angles, fsize, hipMemcpyHostToDevice);

        glm::vec4 algomas(1,2,3,4);
        hipMemcpyToSymbol(HIP_SYMBOL(plane1), algomas, sizeof(glm::vec4), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(plane2), algomas, sizeof(glm::vec4), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(plane3), algomas, sizeof(glm::vec4), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(plane4), algomas, sizeof(glm::vec4), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(plane5), algomas, sizeof(glm::vec4), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(plane6), algomas, sizeof(glm::vec4), 0, hipMemcpyHostToDevice);

        dim3 grid(num_boids,1,1);  // Max 2147483647` , 65535, 65535 blocks
        dim3 block(1,1,1);          // Max 1024 threads per block
        // GPU_update_vector<<<grid,block>>> (
                        // new_updated_directions,
                        // new_directions,
                        // new_positions, num_boids);
        GPU_Update<<<grid,block>>> (
                        d_modelMatrices, new_directions,
                        new_updated_directions, new_positions,
                        new_raxis, new_angles, num_boids, cT);

        hipMemcpy(modelMatrices, d_modelMatrices, m4size, hipMemcpyDeviceToHost);
        hipMemcpy(directions, new_directions, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(updated_directions, new_updated_directions, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(positions, new_positions, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(raxis, new_raxis, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(angles, new_angles, fsize, hipMemcpyDeviceToHost);

        hipFree(d_modelMatrices);
        hipFree(new_directions);
        hipFree(new_updated_directions);
        hipFree(new_positions);
        hipFree(new_raxis);
        hipFree(new_angles);
}
