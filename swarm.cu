#include "hip/hip_runtime.h"
#define GLM_FORCE_RADIANS
#define FLOCKSIZE 10
#define MIN_COLLISON_AVOIDANCE 10
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/norm.hpp>
#include <stdio.h>

#include "main.h"

// __device__ Functions

__device__ int GPU_globalindex(){
        return  blockIdx.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.x * blockDim.z * blockDim.y * blockDim.x +
                threadIdx.z * blockDim.y * blockDim.x +
                threadIdx.y * blockDim.x +
                threadIdx.x;
}

// _global_ Functions

// __global__ void GPU_hashmap(glm::vec3 *a, int n) {
//         int i = GPU_globalindex();
//         if(i < n)
//         {
//                 // a[i].x = int(a[i].x)+9;
//                 // a[i].y = int(a[i].y)+9;
//                 // a[i].z = int(a[i].z)+9;

//                 a[i].x *= 1.05;
//                 a[i].y *= 1.05;
//                 a[i].z *= 1.05;
//         }
// }

typedef struct
{
  float x, y, z;
} point;

__device__ float distance(glm::vec3 a, glm::vec3 b){
  return sqrtf( powf(a.x - b.x,2) + powf(a.y - b.y, 2) + powf(a.z - b.z, 2));
}

__device__ glm::vec3 substract(glm::vec3 a, glm::vec3 b){
  glm::vec3 rpta;
  rpta.x = a.x - b.x;
  rpta.y = a.y - b.y;
  rpta.z = a.z - b.z;
  return rpta;
}

__device__ glm::vec3 direction(glm::vec3 from, glm::vec3 to){
  return substract(to, from);
}

__global__ void GPU_hashmap(glm::vec3 *a, glm::vec3 *b, float *c, int n) {
        int index = GPU_globalindex();
        printf("en kernel");
        if(index < n)
        {
          //printf("Hello thread %d\n", index);

          //////////////////////////////////////////////////////////////////////////////////////////
          ////////// a es Direccion, b es posicion, esa era la idea, cambiale los nombres si gustas, c era 'angulo'
          //////////////////////////////////////////////////////////////////////////////////////////



                // a[index].x *= 1.05;
                // a[index].y *= 1.05;
                // a[index].z *= 1.05;
                int i;
                int cur_i = 0;
                glm::vec3* points = new glm::vec3[n];

                for(i=0;i<n;++i){
                  if(i != index)
                    if(distance(b[index], b[i]) < FLOCKSIZE){
                      points[cur_i] = b[i];
                      cur_i++;
                    }
                }
                printf("cur_i: %d, distance %f\n", cur_i, distance(b[index], b[i]));
                //printf("distance: %f\n", distance(b[index], b[i]));
                // printf("pos_actual: %f %f %f\n", b[index].x, b[index].y, b[index].z);

                for(i=0;i<cur_i;++i){
                  double dist = distance(b[index], points[i]);
                  if(dist < MIN_COLLISON_AVOIDANCE){
                    glm::vec3 temp = substract(b[index], direction(b[index], points[i]));
                    //printf("Hello thread %d %d %d\n", temp.x, temp.y, temp.z);
                    a[index].x = temp.x;
                    a[index].y = temp.y;
                    a[index].z = temp.z;
                  }
                }
        }
}

// __global__ void GPU_inverse(float *a, int n) {
//         int i = GPU_globalindex();
//         if(i < n)
//                 a[i] = 255-a[i];
// }

// __global__ void GPU_grayscale(pix_t *a, pix_t *b, int n, int c) {
//         int i = GPU_globalindex();

//         if(i < n)
//         {
//                 i *= c;
//                 int grey_value = 0;

//                 for(int j=0; j<c; j++)
//                         grey_value += a[i+j];
//                 grey_value /= c;

//                 for(int j=0; j<c; j++)
//                         b[i+j] = grey_value;
//         }
// }

// __global__ void GPU_binarize(pix_t *a, int n, pix_t thresh) {
//         int i = GPU_globalindex();
//         if(i < n)
//                 a[i] = (a[i] >= thresh)*255;
// }

// main functions

// void update(glm::vec3 *a, int n) {
//         glm::vec3 *d_a;
//         size_t size = n * sizeof(glm::vec3);
//         dim3 grid(n,1,1);           // Max 2147483647 , 65535, 65535 blocks
//         dim3 block(1,1,1);          // Max 1024 threads per block

//         hipMalloc(&d_a, size);
//         hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
//         GPU_hashmap<<<grid,block>>> (d_a, n);
//         hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
//         hipFree(d_a);
// }

void update(glm::vec3 *a, glm::vec3 *b, float *c, int n) {
        printf("calling UPDATE\n");
        glm::vec3 *d_a;
        glm::vec3 *d_b;
        float *d_c;

        size_t size = n * sizeof(glm::vec3);

        hipMalloc(&d_a, size);
        hipMalloc(&d_b, size);
        hipMalloc(&d_c, n * sizeof(float));

        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        hipMemcpy(d_c, c, n * sizeof(float), hipMemcpyHostToDevice);

        dim3 grid(n,1,1);           // Max 2147483647 , 65535, 65535 blocks
        dim3 block(1,1,1);          // Max 1024 threads per block
        GPU_hashmap<<<grid,block>>> (d_a, d_b, d_c, n);

        hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
        hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
        hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
}
