#include "hip/hip_runtime.h"
#define GLM_FORCE_RADIANS

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/norm.hpp>

#include "main.h"

const float FLOCKSIZE = 5.0;
#define ATTRACTION_VELOCITY 1.0f
#define ORIENTATION_VELOCITY 0.5f
const float epsilon = 1.0e-4;

// __device__ Functions

__device__ int GPU_globalindex(){
        return  blockIdx.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.x * blockDim.z * blockDim.y * blockDim.x +
                threadIdx.z * blockDim.y * blockDim.x +
                threadIdx.y * blockDim.x +
                threadIdx.x;
}

__device__ void closest_neighbors(int*& points_indices, int& n_points, int global_index, int total_number, glm::vec3 *positions){
  n_points = 0;
  for(int j=0;j<total_number;++j){
    if(j != global_index){
      if(glm::distance(positions[j], positions[global_index]) < FLOCKSIZE){
        points_indices[n_points] = j;
        n_points++;
      }
    }
  }
}

// __global__ Functions

__device__ glm::vec3 resultant(glm::vec3* positions, int* points_indices, int n_points, int cur_boid_index){
    glm::vec3 new_vec(0,0,0);
    for(int j=0;j<n_points;++j)
        new_vec += glm::normalize(positions[points_indices[j]] - positions[cur_boid_index]);
    return new_vec;
}

__device__ glm::vec3 resultant_direction(glm::vec3* directions, int* points_indices, int n_points, int cur_boid_index){
    glm::vec3 new_vec(0,0,0);
    for(int j=0;j<n_points;++j)
        new_vec += glm::normalize(directions[points_indices[j]]);

    return new_vec;
}

__global__ void GPU_update_vector(glm::vec3 *directions_output, glm::vec3 *directions_input, glm::vec3 *positions, int num_boids){
        int index = GPU_globalindex();
        if(index < num_boids)
        {
                directions_output[index] = glm::vec3(0,0,0);
                int n_points = 0;
                int* points_indices = new int[num_boids];
                closest_neighbors(points_indices, n_points, index, num_boids, positions);
                glm::vec3 sum_vector = resultant(positions, points_indices, n_points, index);
                glm::vec3 sum_direction = resultant_direction(directions_input, points_indices, n_points, index);
                directions_output[index] += sum_vector*-1.0f;
                directions_output[index] += sum_vector*ATTRACTION_VELOCITY + sum_direction*ORIENTATION_VELOCITY;
                delete[] points_indices;
        }
}

__global__ void GPU_Update(glm::mat4 *modelMatrices, glm::vec3 *directions, glm::vec3 *updated_directions, glm::vec3 *positions, glm::vec3 *raxis, float *angles, int num_boids, float cT) {
        int i = GPU_globalindex();
        if(i < num_boids)
        {

                float theta = 0.0;
                glm::vec3 cr(0,0,0);
                if(glm::length(directions[i]-updated_directions[i]) > epsilon)
                {
                        theta = glm::acos(glm::dot(glm::normalize(directions[i]),glm::normalize(updated_directions[i])));
                        cr = glm::normalize(glm::cross(directions[i],updated_directions[i]));
                }

                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], -angles[i], raxis[i]);
                }
                modelMatrices[i] = glm::translate(modelMatrices[i],updated_directions[i]*0.0125f);               // Falta Delta, reemp. por 0.0125
                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], angles[i], raxis[i]);
                }


                if(glm::length(cr) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], theta, cr);
                        raxis[i] = glm::normalize(glm::cross(glm::vec3(0.0,0.0,0.25),updated_directions[i]));
                        angles[i] = glm::acos(glm::dot(glm::normalize(updated_directions[i]),glm::normalize(glm::vec3(0.0,0.0,0.25))));;
                }

                directions[i] = updated_directions[i];
                positions[i] += directions[i]*0.0125f;

        }
}

void update(glm::mat4 *modelMatrices, glm::vec3 *directions,
            glm::vec3 *updated_directions, glm::vec3 *positions,
            glm::vec3 *raxis, float *angles, int num_boids, float cT) {
        glm::mat4 *d_modelMatrices;
        glm::vec3 *new_directions, *new_updated_directions, *new_positions, *new_raxis;
        float *new_angles;

        size_t m4size = num_boids * sizeof(glm::mat4);
        size_t v3size = num_boids * sizeof(glm::vec3);
        size_t fsize  = num_boids * sizeof(float);

        hipMalloc(&d_modelMatrices, m4size);
        hipMalloc(&new_directions, v3size);
        hipMalloc(&new_updated_directions, v3size);
        hipMalloc(&new_positions, v3size);
        hipMalloc(&new_raxis, v3size);
        hipMalloc(&new_angles, fsize);

        hipMemcpy(d_modelMatrices, modelMatrices, m4size, hipMemcpyHostToDevice);
        hipMemcpy(new_directions, directions, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_updated_directions, updated_directions, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_positions, positions, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_raxis, raxis, v3size, hipMemcpyHostToDevice);
        hipMemcpy(new_angles, angles, fsize, hipMemcpyHostToDevice);

        dim3 grid(num_boids,1,1);  // Max 2147483647 , 65535, 65535 blocks
        dim3 block(1,1,1);          // Max 1024 threads per block
        GPU_update_vector<<<grid,block>>> (
                        new_updated_directions,
                        new_directions,
                        new_positions, num_boids);
        GPU_Update<<<grid,block>>> (
                        d_modelMatrices, new_directions,
                        new_updated_directions, new_positions,
                        new_raxis, new_angles, num_boids, cT);

        hipMemcpy(modelMatrices, d_modelMatrices, m4size, hipMemcpyDeviceToHost);
        hipMemcpy(directions, new_directions, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(updated_directions, new_updated_directions, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(positions, new_positions, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(raxis, new_raxis, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(angles, new_angles, fsize, hipMemcpyDeviceToHost);

        hipFree(d_modelMatrices);
        hipFree(new_directions);
        hipFree(new_updated_directions);
        hipFree(new_positions);
        hipFree(new_raxis);
        hipFree(new_angles);
}
