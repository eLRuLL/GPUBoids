#include "hip/hip_runtime.h"
#define GLM_FORCE_RADIANS
#include <iostream>
#include <stdio.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/norm.hpp>

#include "main.h"

// __device__ Functions

__device__ int GPU_globalindex(){
        return  blockIdx.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x + 
                blockIdx.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x + 
                blockIdx.x * blockDim.z * blockDim.y * blockDim.x + 
                threadIdx.z * blockDim.y * blockDim.x +
                threadIdx.y * blockDim.x + 
                threadIdx.x;
}

// _global_ Functions

__global__ void GPU_Update(glm::mat4 *modelMatrices, glm::vec3 *d, glm::vec3 *dj, glm::vec3 *c, glm::vec3 *raxis, float *w, int n, float cT) {
        int i = GPU_globalindex();
        if(i < n)
        {

                float epsilon = 1.0e-4;

                dj[i].x = -sinf(cT);
                dj[i].y = 0.00;
                dj[i].z = cosf(cT);

                float theta = 0.0;
                glm::vec3 cr(0,0,0);
                if(glm::length(d[i]-dj[i]) > epsilon)
                {
                        theta = glm::acos(glm::dot(glm::normalize(d[i]),glm::normalize(dj[i])));
                        cr = glm::normalize(glm::cross(d[i],dj[i]));
                }

                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], -w[i], raxis[i]);
                }
                modelMatrices[i] = glm::translate(modelMatrices[i],dj[i]*0.0125f);               // Falta Delta, reemp. por 0.0125
                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], w[i], raxis[i]);
                }


                if(glm::length(cr) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], theta, cr);
                        raxis[i] = glm::normalize(glm::cross(glm::vec3(0.0,0.0,0.25),dj[i]));
                        w[i] = glm::acos(glm::dot(glm::normalize(dj[i]),glm::normalize(glm::vec3(0.0,0.0,0.25))));;
                }

                d[i] = dj[i];
                c[i] += d[i]*0.0125f;

        }
}

void update(glm::mat4 *modelMatrices, glm::vec3 *d, glm::vec3 *dj, glm::vec3 *c, glm::vec3 *raxis, float *w, int n, float cT) {
        glm::mat4 *d_modelMatrices;
        glm::vec3 *d_d, *d_dj, *d_c, *d_raxis;
        float *d_w;

        size_t m4size = n * sizeof(glm::mat4);
        size_t v3size = n * sizeof(glm::vec3);
        size_t fsize = n * sizeof(float);
        
        hipMalloc(&d_modelMatrices, m4size);
        hipMalloc(&d_d, v3size);
        hipMalloc(&d_dj, v3size);
        hipMalloc(&d_c, v3size);
        hipMalloc(&d_raxis, v3size);
        hipMalloc(&d_w, fsize);

        hipMemcpy(d_modelMatrices, modelMatrices, m4size, hipMemcpyHostToDevice);
        hipMemcpy(d_d, d, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_dj, dj, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_c, c, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_raxis, raxis, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_w, w, fsize, hipMemcpyHostToDevice);

        dim3 grid(n,1,1);           // Max 2147483647 , 65535, 65535 blocks
        dim3 block(1,1,1);          // Max 1024 threads per block
        GPU_Update<<<grid,block>>> (d_modelMatrices, d_d, d_dj, d_c, d_raxis, d_w, n, cT);
        
        hipMemcpy(modelMatrices, d_modelMatrices, m4size, hipMemcpyDeviceToHost);
        hipMemcpy(d, d_d, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(dj, d_dj, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(c, d_c, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(raxis, d_raxis, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(w, d_w, fsize, hipMemcpyDeviceToHost);
        
        hipFree(d_modelMatrices);
        hipFree(d_d);
        hipFree(d_dj);
        hipFree(d_c);
        hipFree(d_raxis);
        hipFree(d_w);
}