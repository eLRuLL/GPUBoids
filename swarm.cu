#include "hip/hip_runtime.h"
#define GLM_FORCE_RADIANS
// #define FLOCKSIZE 10
// #define MIN_COLLISON_AVOIDANCE 10
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/norm.hpp>

#include "main.h"

const float FLOCKSIZE = 5.0;
#define ATTRACTION_VELOCITY 1.0f
#define ORIENTATION_VELOCITY 0.5f
const float epsilon = 1.0e-4;

// __device__ Functions

__device__ int GPU_globalindex(){
        return  blockIdx.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                blockIdx.x * blockDim.z * blockDim.y * blockDim.x +
                threadIdx.z * blockDim.y * blockDim.x +
                threadIdx.y * blockDim.x +
                threadIdx.x;
}

__device__ void closest_neighbors(int*& points_indices, int& n_points, int global_index, int total_number, glm::vec3 *positions){
  n_points = 0;
  for(int j=0;j<total_number;++j){
    if(j != global_index){
      if(glm::distance(positions[j], positions[global_index]) < FLOCKSIZE){
        points_indices[n_points] = j;
        n_points++;
      }
    }
  }
}

// __global__ Functions

__device__ glm::vec3 resultant(glm::vec3* positions, int* points_indices, int n_points, int cur_boid_index){
    glm::vec3 new_vec(0,0,0);
    for(int j=0;j<n_points;++j)
        new_vec += glm::normalize(positions[points_indices[j]] - positions[cur_boid_index]);
    return new_vec;
}

__device__ glm::vec3 resultant_direction(glm::vec3* directions, int* points_indices, int n_points, int cur_boid_index){
    glm::vec3 new_vec(0,0,0);
    for(int j=0;j<n_points;++j)
        new_vec += glm::normalize(directions[points_indices[j]]);

    return new_vec;
}

__global__ void GPU_update_vector(glm::vec3 *directions_output, glm::vec3 *directions_input, glm::vec3 *positions, int n){
        int index = GPU_globalindex();
        if(index < n)
        {
                directions_output[index] = glm::vec3(0,0,0);
                int n_points = 0;
                int* points_indices = new int[n];
                closest_neighbors(points_indices, n_points, index, n, positions);
                glm::vec3 sum_vector = resultant(positions, points_indices, n_points, index);
                glm::vec3 sum_direction = resultant_direction(directions_input, points_indices, n_points, index);
                directions_output[index] += sum_vector*-1.0f;
                directions_output[index] += sum_vector*ATTRACTION_VELOCITY + sum_direction*ORIENTATION_VELOCITY;
                delete[] points_indices;
        }
}

__global__ void GPU_Update(glm::mat4 *modelMatrices, glm::vec3 *d, glm::vec3 *dj, glm::vec3 *c, glm::vec3 *raxis, float *w, int n, float cT) {
        int i = GPU_globalindex();
        if(i < n)
        {

                float theta = 0.0;
                glm::vec3 cr(0,0,0);
                if(glm::length(d[i]-dj[i]) > epsilon)
                {
                        theta = glm::acos(glm::dot(glm::normalize(d[i]),glm::normalize(dj[i])));
                        cr = glm::normalize(glm::cross(d[i],dj[i]));
                }

                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], -w[i], raxis[i]);
                }
                modelMatrices[i] = glm::translate(modelMatrices[i],glm::normalize(dj[i])*0.0125f);
                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], w[i], raxis[i]);
                }


                if(glm::length(cr) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], theta, cr);
                        raxis[i] = glm::normalize(glm::cross(glm::vec3(0.0,0.0,0.25),dj[i]));
                        w[i] = glm::acos(glm::dot(glm::normalize(dj[i]),glm::normalize(glm::vec3(0.0,0.0,0.25))));;
                }

                d[i] = dj[i];
                c[i] += d[i]*0.0125f;

        }
}

void update(glm::mat4 *modelMatrices, glm::vec3 *d, glm::vec3 *dj, glm::vec3 *c, glm::vec3 *raxis, float *w, int n, float cT) {

        glm::mat4 *d_modelMatrices;
        glm::vec3 *d_d, *d_dj, *d_c, *d_raxis;
        float *d_w;

        size_t m4size = n * sizeof(glm::mat4);
        size_t v3size = n * sizeof(glm::vec3);
        size_t fsize = n * sizeof(float);

        hipMalloc(&d_modelMatrices, m4size);
        hipMalloc(&d_d, v3size);
        hipMalloc(&d_dj, v3size);
        hipMalloc(&d_c, v3size);
        hipMalloc(&d_raxis, v3size);
        hipMalloc(&d_w, fsize);

        hipMemcpy(d_modelMatrices, modelMatrices, m4size, hipMemcpyHostToDevice);
        hipMemcpy(d_d, d, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_dj, dj, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_c, c, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_raxis, raxis, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_w, w, fsize, hipMemcpyHostToDevice);

        dim3 grid(n,1,1);           // Max 2147483647 , 65535, 65535 blocks
        dim3 block(1,1,1);          // Max 1024 threads per block
        GPU_update_vector<<<grid,block>>> (d_dj, d_d, d_c,n);
        GPU_Update<<<grid,block>>> (d_modelMatrices, d_d, d_dj, d_c, d_raxis, d_w, n, cT);

        hipMemcpy(modelMatrices, d_modelMatrices, m4size, hipMemcpyDeviceToHost);
        hipMemcpy(d, d_d, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(dj, d_dj, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(c, d_c, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(raxis, d_raxis, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(w, d_w, fsize, hipMemcpyDeviceToHost);

        hipFree(d_modelMatrices);
        hipFree(d_d);
        hipFree(d_dj);
        hipFree(d_c);
        hipFree(d_raxis);
        hipFree(d_w);
}
