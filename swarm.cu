#include "hip/hip_runtime.h"
#define GLM_FORCE_RADIANS
// #define FLOCKSIZE 10
// #define MIN_COLLISON_AVOIDANCE 10
#include <iostream>
#include <stdio.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/norm.hpp>

#include "main.h"

const float FLOCKSIZE = 3.0;
const float MIN_COLLISON_AVOIDANCE = 3.0;
const float epsilon = 1.0e-4;

// __device__ Functions

__device__ int GPU_globalindex(){
        return  blockIdx.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x + 
                blockIdx.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x + 
                blockIdx.x * blockDim.z * blockDim.y * blockDim.x + 
                threadIdx.z * blockDim.y * blockDim.x +
                threadIdx.y * blockDim.x + 
                threadIdx.x;
}

// __global__ Functions

__global__ void GPU_Repel(glm::vec3 *dj, glm::vec3 *c, int n){
        int i = GPU_globalindex();
        if(i < n)
        {

                int j;
                int cur_j = 0;
                glm::vec3* points = new glm::vec3[n];

                for(j=0;j<n;++j){
                  if(j != i)
                    if(glm::distance(c[j], c[i]) < FLOCKSIZE){
                      points[cur_j] = c[j];
                      cur_j++;
                    }
                }

                for(j=0;j<cur_j;++j){
                  float dist = glm::distance(c[i], points[j]);
                  if(dist < MIN_COLLISON_AVOIDANCE){
                    dj[i] += glm::normalize(points[j] - c[i])*(-1.0f);
                    
                  }
                }
                delete[] points;
        }
}

__global__ void GPU_Update(glm::mat4 *modelMatrices, glm::vec3 *d, glm::vec3 *dj, glm::vec3 *c, glm::vec3 *raxis, float *w, int n, float cT) {
        int i = GPU_globalindex();
        if(i < n)
        {

                float theta = 0.0;
                glm::vec3 cr(0,0,0);
                if(glm::length(d[i]-dj[i]) > epsilon)
                {
                        theta = glm::acos(glm::dot(glm::normalize(d[i]),glm::normalize(dj[i])));
                        cr = glm::normalize(glm::cross(d[i],dj[i]));
                }

                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], -w[i], raxis[i]);
                }
                modelMatrices[i] = glm::translate(modelMatrices[i],dj[i]*0.0125f);               // Falta Delta, reemp. por 0.0125
                if(glm::length(raxis[i]) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], w[i], raxis[i]);
                }


                if(glm::length(cr) > epsilon)
                {
                        modelMatrices[i] = glm::rotate(modelMatrices[i], theta, cr);
                        raxis[i] = glm::normalize(glm::cross(glm::vec3(0.0,0.0,0.25),dj[i]));
                        w[i] = glm::acos(glm::dot(glm::normalize(dj[i]),glm::normalize(glm::vec3(0.0,0.0,0.25))));;
                }

                d[i] = dj[i];
                c[i] += d[i]*0.0125f;

        }
}

void update(glm::mat4 *modelMatrices, glm::vec3 *d, glm::vec3 *dj, glm::vec3 *c, glm::vec3 *raxis, float *w, int n, float cT) {
        
        glm::mat4 *d_modelMatrices;
        glm::vec3 *d_d, *d_dj, *d_c, *d_raxis;
        float *d_w;

        size_t m4size = n * sizeof(glm::mat4);
        size_t v3size = n * sizeof(glm::vec3);
        size_t fsize = n * sizeof(float);
        
        hipMalloc(&d_modelMatrices, m4size);
        hipMalloc(&d_d, v3size);
        hipMalloc(&d_dj, v3size);
        hipMalloc(&d_c, v3size);
        hipMalloc(&d_raxis, v3size);
        hipMalloc(&d_w, fsize);

        hipMemcpy(d_modelMatrices, modelMatrices, m4size, hipMemcpyHostToDevice);
        hipMemcpy(d_d, d, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_dj, dj, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_c, c, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_raxis, raxis, v3size, hipMemcpyHostToDevice);
        hipMemcpy(d_w, w, fsize, hipMemcpyHostToDevice);

        dim3 grid(n,1,1);           // Max 2147483647 , 65535, 65535 blocks
        dim3 block(1,1,1);          // Max 1024 threads per block
        GPU_Repel<<<grid,block>>> (d_dj, d_c,n);
        GPU_Update<<<grid,block>>> (d_modelMatrices, d_d, d_dj, d_c, d_raxis, d_w, n, cT);
        
        hipMemcpy(modelMatrices, d_modelMatrices, m4size, hipMemcpyDeviceToHost);
        hipMemcpy(d, d_d, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(dj, d_dj, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(c, d_c, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(raxis, d_raxis, v3size, hipMemcpyDeviceToHost);
        hipMemcpy(w, d_w, fsize, hipMemcpyDeviceToHost);
        
        hipFree(d_modelMatrices);
        hipFree(d_d);
        hipFree(d_dj);
        hipFree(d_c);
        hipFree(d_raxis);
        hipFree(d_w);
}